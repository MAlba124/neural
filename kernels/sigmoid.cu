
#include <hip/hip_runtime.h>
extern "C" __global__ void
sigmoid(float *A) {
    A[threadIdx.x] = 1.0 / (1.0 + expf(-A[threadIdx.x]));
}
